#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"
#include <stdio.h>
#include "math_functions.h"
#include <cmath>
#include <stdio.h>

//Variables punto flotante hay perdida de color

#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>

struct sphere
{
	//Origen
	float x;
	float y;
	float z;
	//Sphere
	float radio;
	//material
	uchar r;
	uchar g;
	uchar b;
	float difrac;
	float refrac;
};

struct ray
{
	//Origen
	float x;
	float y;
	float z;
	//NORMALIZADOOOOOO!
	float d_x;
	float d_y;
	float d_z;
};

struct light
{
	//Origen
	float x;
	float y;
	float z;
	//Sphere
	float radio;
	//Color n stuff
	float intensity;
	uchar r;
	uchar g;
	uchar b;
};

struct vector3
{
	float x;
	float y;
	float z;
};

__device__ float dotP(vector3 vec1, vector3 vec2) {
	return vec1.x * vec2.x + vec1.y * vec2.y + vec1.z * vec2.z;
}

__device__ void normalize(vector3* vector) {
	//Normalizar
	float aux = sqrtf(vector->x * vector->x + vector->y * vector->y + vector->z * vector->z);

	vector->x /= aux;
	vector->y /= aux;
	vector->z /= aux;
}

__device__ bool sphereIntersection(ray* ray_test, sphere* obj, float* dist)
{
	float vX, vY, vZ;
	float discrimiante;

	float a, b, c;

	vX = ray_test->x - obj->x;
	vY = ray_test->y - obj->y;
	vZ = ray_test->z - obj->z;

	a = (ray_test->d_x * ray_test->d_x + ray_test->d_y * ray_test->d_y + ray_test->d_z * ray_test->d_z);
	b = 2.0f * (vX * ray_test->d_x + vY * ray_test->d_y + vZ * ray_test->d_z);
	c = (vX * vX + vY * vY + vZ * vZ) - (obj->radio * obj->radio);
	discrimiante = (b * b) - (4 * a * c);
	if (discrimiante < 0.0f)
		return false;
	else
	{
		*dist = (-b - sqrtf(discrimiante)) / (2.0f * a);
		return true;

	}

	return false;

}

__device__ vector3 phongShading(light* luzes, int num_luzes, vector3* point, vector3* normal, vector3* camera, vector3* color)
{
	//Factores de Phong
	float ambiental = 0.2;
	float difuso = 0.5;
	float specular = 0.3f;
	float brillantez = 100;

	vector3 colorSalida;
	colorSalida.x = 0;
	colorSalida.y = 0;
	colorSalida.z = 0;

	// Calcular la contribución de cada luz
	for (int i = 0; i < num_luzes; ++i) {
		light luz = luzes[i];

		// Calcular componente ambiental
		colorSalida.x += color->x * ambiental * luz.intensity;
		colorSalida.y += color->y * ambiental * luz.intensity;
		colorSalida.z += color->z * ambiental * luz.intensity;

		// Calcular componente difusa
		vector3 vec_luz;
		vec_luz.x = luz.x - point->x;
		vec_luz.y = luz.y - point->y;
		vec_luz.z = luz.z - point->z;

		normalize(&vec_luz);

		float doc_prod = dotP(vec_luz, *normal);

		if (doc_prod > 0)
		{
			doc_prod *= difuso * luz.intensity;
			colorSalida.x += doc_prod * color->x;
			colorSalida.y += doc_prod * color->y;
			colorSalida.z += doc_prod * color->z;

			// Calcular componente especular
			vector3 rVect;
			rVect.x = vec_luz.x - 2.0f * (doc_prod)*normal->x;
			rVect.y = vec_luz.y - 2.0f * (doc_prod)*normal->y;
			rVect.z = vec_luz.z - 2.0f * (doc_prod)*normal->z;

			vector3 dirCam;
			dirCam.x = camera->x - point->x;
			dirCam.y = camera->y - point->y;
			dirCam.z = camera->z - point->z;

			normalize(&dirCam);
			//Necesitamos vector que va a la camara. Calculamos el vector que va a la camara.
			float dotVR = dotP(rVect, dirCam);
			dotVR *= powf(dotVR, brillantez) * specular * luz.intensity;

			colorSalida.x += dotVR * color->x;
			colorSalida.y += dotVR * color->y;
			colorSalida.z += dotVR * color->z;
		}
	}

	colorSalida.x = min(255, (int)roundf(colorSalida.x));
	colorSalida.y = min(255, (int)roundf(colorSalida.y));
	colorSalida.z = min(255, (int)roundf(colorSalida.z));

	return colorSalida;
}


__global__ void rayCasting(vector3* camera, light* luz, vector3* pi_corner, uchar* output, sphere* objects, int num_esferas, int num_luz, int width, int heigth, float inc_x, float inc_y)
{
	//columna
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//fila
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (i < width && j < heigth)
	{
		int idx = j * width * 3 + i * 3;

		ray primary;
		// Origen de rayos
		primary.x = camera->x;
		primary.y = camera->y;
		primary.z = camera->z;

		// Ver la esquina izquierda de cada pixel para cada thread
		vector3 dest;
		dest.x = pi_corner->x + inc_x * i;
		dest.y = pi_corner->y - inc_y * j;
		dest.z = 1;

		// Rayo vector de dirección sin normalizar
		primary.d_x = dest.x - primary.x;
		primary.d_y = dest.y - primary.y;
		primary.d_z = dest.z - primary.z;
		// Normalizar rayos
		float aux = sqrtf(primary.d_x * primary.d_x + primary.d_y * primary.d_y + primary.d_z * primary.d_z);
		primary.d_x /= aux;
		primary.d_y /= aux;
		primary.d_z /= aux;

		//llenamos de un color
		float distance = 0; // Distancia para ver donde esta el choque. 
		//Foxu 3 spheres
		float min_dist = 1000;
		vector3 hit_point;
		sphere hit_sphere;
		bool intersected = false; //Si choco con algo es el interPoint
		//Foxu 3 spheres
		// Iterar sobre todas las esferas para encontrar la más cercana
		for (int k = 0; k < num_esferas; k++) {
			if (sphereIntersection(&primary, &objects[k], &distance)) {
				intersected = true;
				if (distance < min_dist) {
					min_dist = distance;
					hit_point.x = primary.d_x * distance + primary.x;
					hit_point.y = primary.d_y * distance + primary.y;
					hit_point.z = primary.d_z * distance + primary.z;
					hit_sphere = objects[k]; //Con que esfera choque. 
				}
			}
		}

		if (intersected) {
			// Calcular vector normal de la intersección del rayo con la esfera.
			vector3 normal;
			normal.x = hit_point.x - hit_sphere.x;
			normal.y = hit_point.y - hit_sphere.y;
			normal.z = hit_point.z - hit_sphere.z;
			// Normalizar el vector
			normalize(&normal);

			// Color de la esferas
			vector3 colorInicio;
			colorInicio.x = hit_sphere.r;
			colorInicio.y = hit_sphere.g;
			colorInicio.z = hit_sphere.b;

			vector3 cameraVect;
			cameraVect.x = camera->x - hit_point.x;
			cameraVect.y = camera->y - hit_point.y;
			cameraVect.z = camera->z - hit_point.z;

			normalize(&cameraVect);

			colorInicio = phongShading(luz, num_luz, &hit_point, &normal, &cameraVect, &colorInicio);
			//Generamos el reflect
			//Vemos si reflect choca con algo
			// Si choca le pasamos colorInicio y le ajustamos el color.
			// Si no choca le ponemos directo colorInicio.
			//

			output[idx] = colorInicio.z; // Blue
			output[idx + 1] = colorInicio.y; // Green
			output[idx + 2] = colorInicio.x; // Red
		}
		else {
			output[idx] = 30;
			output[idx + 1] = 30;
			output[idx + 2] = 30;
		}

	}

}


int main()
{
	//creamos camara
	vector3 camera;
	camera.x = 0;
	camera.y = 0;
	camera.z = 0;

	//Tamano de imagen en pixeles y tamano de plano
	int width = 500, height = 500;
	float tam_imgX = 2, tam_imgY = 2;

	int num_esferas = 3;
	sphere esferas_host[3];
	//Morada
	esferas_host[0].x = 0;
	esferas_host[0].y = 0;
	esferas_host[0].z = 20;
	esferas_host[0].r = 255;
	esferas_host[0].g = 0;
	esferas_host[0].b = 158;
	esferas_host[0].radio = 2;
	//Verde claro
	esferas_host[1].x = 0;
	esferas_host[1].y = -5;
	esferas_host[1].z = 10;
	esferas_host[1].r = 50;
	esferas_host[1].g = 168;
	esferas_host[1].b = 123;
	esferas_host[1].radio = 2;
	//Naranja
	esferas_host[2].x = 0;
	esferas_host[2].y = 5;
	esferas_host[2].z = 10;
	esferas_host[2].r = 200;
	esferas_host[2].g = 100;
	esferas_host[2].b = 50;
	esferas_host[2].radio = 2;
	//////////////////////

	//Creamos una luz en el mundo
	int num_luz = 3;
	light light_host[2];
	light_host[0].x = 25;
	light_host[0].y = 3;
	light_host[0].z = 1;
	light_host[0].radio = 1;
	light_host[0].intensity = 0.5;
	light_host[1].x = 10;
	light_host[1].y = 3;
	light_host[1].z = -1;
	light_host[1].radio = 1;
	light_host[1].intensity = 1;
	//light luz1;
	//luz1.x = 25;
	//luz1.y = 3;
	//luz1.z = 1;
	//luz1.radio = 1;
	//// Segunda Luz
	//light luz2;
	//luz2.x = 5;
	//luz2.y = 3;
	//luz2.z = -1;
	//luz2.radio = 1;


	//esquina superior izquierda de la pantalla
	vector3 esquina_img;
	esquina_img.x = -1;
	esquina_img.y = tam_imgY / 2.0f;
	esquina_img.z = 1;

	/*Calcular el tamano de cuanto va a medir cuanto pixel en el espacio 3d. (Pasar los 500x500 pixeles a los 2x2 del mundo) es el valor de cuanto en cuanto me
	voy moviendo de la esquina*/
	float inc_x = tam_imgX / width;
	float inc_y = tam_imgY / height;

	//agregamos desf al cent
	esquina_img.y -= inc_y / 2.0f;
	esquina_img.z += inc_x / 2.0f;

	//Pasar memoria de cuda
	vector3* camera_dev;
	sphere* esferas_dev;
	vector3* esquina_dev;
	//Imagen destino donde guardar
	uchar* img_dev;
	//Luz destino
	//light* luz_dev;
	//light* luz_dev2;
	light* lights_dev;

	dim3 threads(16, 16);
	dim3 blocks(ceil((float)width / (float)threads.x), ceil((float)height / (float)threads.y));

	hipMalloc(&camera_dev, sizeof(vector3));
	hipMalloc(&img_dev, width * height * 3);
	hipMalloc(&esferas_dev, 3 * sizeof(sphere));
	hipMalloc(&esquina_dev, sizeof(vector3));
	hipMalloc(&lights_dev, 2 * sizeof(light));
	//hipMalloc(&luz_dev, sizeof(light));
	//hipMalloc(&luz_dev2, sizeof(light));
	// Copiar datos del host al dispositivo

	hipMemcpy(camera_dev, &camera, sizeof(vector3), hipMemcpyHostToDevice);
	//hipMemcpy(esferas_dev, &esf1, sizeof(esfera), hipMemcpyHostToDevice);
	hipMemcpy(esferas_dev, esferas_host, 3 * sizeof(sphere), hipMemcpyHostToDevice);
	hipMemcpy(esquina_dev, &esquina_img, sizeof(vector3), hipMemcpyHostToDevice);
	/*hipMemcpy(luz_dev, &luz1, sizeof(light), hipMemcpyHostToDevice);
	hipMemcpy(luz_dev2, &luz2, sizeof(light), hipMemcpyHostToDevice);*/
	hipMemcpy(lights_dev, light_host, 2 * sizeof(light), hipMemcpyHostToDevice);
	rayCasting << <blocks, threads >> > (camera_dev, lights_dev, esquina_dev, img_dev, esferas_dev, num_esferas, num_luz, width, height, inc_x, inc_y);

	cv::Mat frame = cv::Mat(cv::Size(width, height), CV_8UC3);
	hipMemcpy(frame.ptr(), img_dev, width * height * 3, hipMemcpyDeviceToHost);

	cv::imshow("salida", frame);
	cv::waitKey(0);

	// Liberar memoria en el dispositivo CUDA
	hipFree(camera_dev);
	hipFree(img_dev);
	hipFree(esferas_dev);
	hipFree(esquina_dev);
	hipFree(lights_dev);

	return 0;
}